#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"
#include "support.h"

const unsigned int numStream = 3;

int main (int argc, char *argv[])
{

    Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    float *A_h, *B_h, *C_h;

    float *A_d, *B_d, *C_d;

    size_t A_sz, B_sz, C_sz;
    unsigned matArow, matAcol;
    unsigned matBrow, matBcol;
    dim3 dim_grid, dim_block;
    unsigned VecSize;



    hipStream_t streams[numStream];
    for (int i = 0; i < numStream; i++)
        hipStreamCreate(&streams[i]);


    if (argc == 1) {
        matArow = 1000;
        matAcol = matBrow = 1000;
        matBcol = 1000;
    } else if (argc == 2) {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[1]);
        matBcol = atoi(argv[1]);
    } else if (argc == 4) {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[2]);
        matBcol = atoi(argv[3]);
    } else {
        printf("\n    Invalid input parameters!"
      "\n    Usage: ./sgemm-tiled                # All matrices are 1000 x 1000"
      "\n    Usage: ./sgemm-tiled <m>            # All matrices are m x m"
      "\n    Usage: ./sgemm-tiled <m> <k> <n>    # A: m x k, B: k x n, C: m x n"
      "\n");
        exit(0);
    }
   
    A_sz = matArow*matAcol;
    B_sz = matBrow*matBcol;
    C_sz = matArow*matBcol;

    VecSize = matArow*matAcol;
    const int segmentLen = VecSize / numStream;

    // A_h = (float*) malloc( sizeof(float)*A_sz );
    hipHostAlloc((void**)&A_h, A_sz*sizeof(float), hipHostMallocDefault);
    for (unsigned int i=0; i < A_sz; i++) { A_h[i] = (rand()%100)/100.00; }
    // hipHostAlloc((void**)&a, A_sz*sizeof(float), hipHostMallocDefault);
    hipHostAlloc((void**)&B_h, B_sz*sizeof(float), hipHostMallocDefault);
    for (unsigned int i=0; i < A_sz; i++) { B_h[i] = (rand()%100)/100.00; }

    // hipHostAlloc((void**)&b, A_sz*sizeof(float), hipHostMallocDefault);
    // for (unsigned int i=0; i < B_sz; i++) { b[i] = (rand()%100)/100.00; }

    // C_h = (float*) malloc( sizeof(float)*C_sz );
    hipHostAlloc((void**)&C_h, C_sz*sizeof(float), hipHostMallocDefault);
    // hipHostAlloc((void**)&c, A_sz*sizeof(float), hipHostMallocDefault);


    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    A: %u x %u\n    B: %u x %u\n    C: %u x %u\n", matArow, matAcol,
        matBrow, matBcol, matArow, matBcol);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    /*************************************************************************/
    //INSERT CODE HERE


    hipMalloc((float **)&A_d, sizeof(float) * VecSize);
    hipMalloc((float **)&B_d, sizeof(float) * VecSize);
    hipMalloc((float **)&C_d, sizeof(float) * VecSize);

    // hipMallocManaged(&A_d, sizeof(float) * VecSize)
    // hipMallocManaged(&B_d, sizeof(float) * VecSize)
    // hipMallocManaged(&C_d, sizeof(float) * VecSize)


    /*************************************************************************/
	
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------
    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);
	
    /*************************************************************************/

    //INSERT CODE HERE
//   const int blockSize = 256, nStreams = 4;
//   const int n = 4 * 1024 * blockSize * nStreams;
//   const int streamSize = n / nStreams;
//   const int streamBytes = streamSize * sizeof(float);
//     for (int i = 0; i < nStreams; ++i) {
//     int offset = i * streamSize;
//     checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset], 
//                                streamBytes, hipMemcpyHostToDevice, 
//                                stream[i]) );
//     kernel<<<streamSize/blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
//     checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset], 
//                                streamBytes, hipMemcpyDeviceToHost,
//                                stream[i]) );
//   }
    for (int i = 0; i < numStream; i++)
    {   
        // printf("segment..."); fflush(stdout);
        printf("%u heree\n", segmentLen + VecSize % numStream);fflush(stdout);
        printf("%u legth\n", segmentLen );fflush(stdout);
        
        int Offset = i * segmentLen;
        if (i != numStream-1) {
            hipMemcpyAsync(&A_d[Offset], &A_h[Offset], sizeof(float)*segmentLen, hipMemcpyHostToDevice, streams[i]);
            hipMemcpyAsync(B_d, B_h, sizeof(float)*VecSize, hipMemcpyHostToDevice, streams[i]);
            
            basicSgemmStream(matArow,matArow,matArow, &A_d[Offset], B_d, &C_d[Offset], streams[i]);

            hipMemcpyAsync(&C_h[Offset], &C_d[Offset], sizeof(float)*segmentLen, hipMemcpyDeviceToHost, streams[i]);

        }
        else {
            Offset = (i * segmentLen) + VecSize % numStream;
            hipMemcpyAsync(&A_d[Offset], &A_h[Offset], sizeof(float)*(segmentLen+ VecSize % numStream), hipMemcpyHostToDevice, streams[i]);
            hipMemcpyAsync(B_d, B_h, sizeof(float)*(VecSize), hipMemcpyHostToDevice, streams[i]);
            
            basicSgemmStream(matArow,matArow,matArow, &A_d[Offset], B_d, &C_d[Offset], streams[i]);
            
            hipMemcpyAsync(&C_h[Offset], &C_d[Offset], sizeof(float)*(segmentLen + VecSize % numStream), hipMemcpyDeviceToHost, streams[i]);

        }

        hipStreamSynchronize(streams[i]);
    }

    /*************************************************************************/
    
    // hipDeviceSynchronize();
    // stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel using standard sgemm interface ---------------------------
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);


    // cuda_ret = hipDeviceSynchronize();
    // if(cuda_ret != hipSuccess) printf("Unable to launch kernel");
    // stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------
    printf("Copying data from device to host..."); fflush(stdout);
    // startTime(&timer);

    /*************************************************************************/
    //INSERT CODE HERE
    // hipMemcpy(C_h, C_d, sizeof(float) * C_sz, hipMemcpyDeviceToHost);	

    /*************************************************************************/

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    // printf(C_h, "c_h");fflush(stdout);

    verify(A_h, B_h, C_h, matArow, matAcol, matBcol);


    // Free memory ------------------------------------------------------------
    // free(A_h);
    // free(B_h);
    // free(C_h);

    hipHostFree(A_h);
    hipHostFree(B_h);
    hipHostFree(C_h);
    /*************************************************************************/
    //INSERT CODE HERE
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    for (int i = 0; i < numStream; i++)
    {
        hipStreamDestroy(streams[i]);
    }
    /*************************************************************************/
;
    return 0;
}

