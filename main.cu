#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"
#include "support.h"

const unsigned int numStream = 3;

int main (int argc, char *argv[])
{

    Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    float *A_h, *B_h, *C_h;

    float *A_d, *B_d, *C_d;

    float *A_ds[numStream], *B_ds[numStream], *C_ds[numStream];



    // float *hA1,*hA2,*hB1,*hB2,*hC1,*hC2,*hC3,*hC4;
    // float *dA1,*dA1_2,*dA2,*dA2_2,*dB1,*dB1_2,*dB2,*dB2_2;
    // float *dC1,*dC2,*dC3,*dC4;



    size_t A_sz, B_sz, C_sz;
    unsigned matArow, matAcol;
    unsigned matBrow, matBcol;
    dim3 dim_grid, dim_block;
    unsigned VecSize;



    hipStream_t streams[numStream];
    for (int i = 0; i < numStream; i++)
        hipStreamCreate(&streams[i]);


    if (argc == 1) {
        matArow = 1000;
        matAcol = matBrow = 1000;
        matBcol = 1000;
    } else if (argc == 2) {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[1]);
        matBcol = atoi(argv[1]);
    } else if (argc == 4) {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[2]);
        matBcol = atoi(argv[3]);
    } else {
        printf("\n    Invalid input parameters!"
      "\n    Usage: ./sgemm-tiled                # All matrices are 1000 x 1000"
      "\n    Usage: ./sgemm-tiled <m>            # All matrices are m x m"
      "\n    Usage: ./sgemm-tiled <m> <k> <n>    # A: m x k, B: k x n, C: m x n"
      "\n");
        exit(0);
    }
   
    A_sz = matArow*matAcol;
    B_sz = matBrow*matBcol;
    C_sz = matArow*matBcol;

    VecSize = matArow*matAcol;
    const int segmentLen = VecSize / numStream;

    // A_h = (float*) malloc( sizeof(float)*A_sz );
    hipHostAlloc((void**)&A_h, A_sz*sizeof(float), hipHostMallocDefault);
    for (unsigned int i=0; i < A_sz; i++) { A_h[i] = (rand()%100)/100.00; }
    // hipHostAlloc((void**)&a, A_sz*sizeof(float), hipHostMallocDefault);
    // for (unsigned int i=0; i < A_sz; i++) { a[i] = (rand()%100)/100.00; }

    // B_h = (float*) malloc( sizeof(float)*B_sz );
    hipHostAlloc((void**)&B_h, B_sz*sizeof(float), hipHostMallocDefault);

    for (unsigned int i=0; i < A_sz; i++) { B_h[i] = (rand()%100)/100.00; }

    // hipHostAlloc((void**)&b, A_sz*sizeof(float), hipHostMallocDefault);
    // for (unsigned int i=0; i < B_sz; i++) { b[i] = (rand()%100)/100.00; }

    // C_h = (float*) malloc( sizeof(float)*C_sz );
    hipHostAlloc((void**)&C_h, C_sz*sizeof(float), hipHostMallocDefault);
    // hipHostAlloc((void**)&c, A_sz*sizeof(float), hipHostMallocDefault);


    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    A: %u x %u\n    B: %u x %u\n    C: %u x %u\n", matArow, matAcol,
        matBrow, matBcol, matArow, matBcol);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    /*************************************************************************/
    //INSERT CODE HERE
    // hipMalloc((float**) &A_d, sizeof(float)*A_sz);
    // hipMalloc((float**) &B_d, sizeof(float)*B_sz);
    // hipMalloc((float**) &C_d, sizeof(float)*C_sz); 

    // for (int i = 0; i < numStream; i++)
    // {
    //     if (i != numStream-1)
    //     {
    //         hipMalloc((void**) &A_d[i], sizeof(float) * segmentLen);
    //         hipMalloc((void**) &B_d[i], sizeof(float) * segmentLen);
    //         hipMalloc((void**) &C_d[i], sizeof(float) * segmentLen);
    //     }
    //     else    // remainder
    //     {
    //         hipMalloc((void**) &A_d[i], sizeof(float) * (segmentLen + VecSize % numStream));
    //         hipMalloc((void**) &B_d[i], sizeof(float) * (segmentLen + VecSize % numStream));
    //         hipMalloc((void**) &C_d[i], sizeof(float) * (segmentLen + VecSize % numStream));
    //     }
    // }

    for (int i = 0; i < numStream; i++)
    {
            hipMalloc((float**) &A_ds[i], sizeof(float) * VecSize);
            hipMalloc((float**) &B_ds[i], sizeof(float) * VecSize);
            hipMalloc((float**) &C_ds[i], sizeof(float) * VecSize);

    }
    hipMalloc((int **)&A_d, sizeof(float) * VecSize);
    hipMalloc((int **)&B_d, sizeof(float) * VecSize);
    hipMalloc((int **)&C_d, sizeof(float) * VecSize);


    /*************************************************************************/
	
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------
    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);
	
    /*************************************************************************/



//     hipStreamSynchronize(streams1); // wait for stream1 to finish
//     hipStreamSynchronize(streams2); 




    //INSERT CODE HERE
    // hipMemcpy(A_d, A_h, sizeof(float) * A_sz, hipMemcpyHostToDevice);
    // hipMemcpy(B_d, B_h, sizeof(float) * B_sz, hipMemcpyHostToDevice);

    // for (int i = 0; i < numStream; i++)
    // {
    //     if (i != numStream-1)
    //     {
    //         hipMemcpyAsync(A_ds[i], A_h, sizeof(float)*segmentLen, hipMemcpyHostToDevice, streams[i]);
    //         hipMemcpyAsync(B_ds[i], B_h , sizeof(float)*segmentLen, hipMemcpyHostToDevice, streams[i]);
    //     }
    //     else
    //     {
    //         hipMemcpyAsync(A_ds[i], A_h , sizeof(float)*(segmentLen + VecSize % numStream), hipMemcpyHostToDevice, streams[i]);
    //         hipMemcpyAsync(B_ds[i], B_h , sizeof(float)*(segmentLen + VecSize % numStream), hipMemcpyHostToDevice, streams[i]);
    //     }
    // }


    for (int i = 0; i < numStream; i++)
    {

        
        int Offset = i * segmentLen;
        // hipMemcpyAsync(&A_d[Offset], &A_h[Offset], sizeof(float)*(segmentLen + VecSize % numStream), hipMemcpyHostToDevice, streams[i]);
        // hipMemcpyAsync(&B_d[Offset], &B_h[Offset], sizeof(float)*(segmentLen + VecSize % numStream), hipMemcpyHostToDevice, streams[i]);
        
        // // basicSgemmStream(matArow/numStream, matArow/numStream, matArow/numStream, A_ds[Offset], B_ds[Offset], C_ds[Offset], streams[i]);
        // basicSgemmStream(matArow,matArow,matArow, &A_d[Offset], B_d, &C_d[Offset], streams[i]);
        if (i != numStream-1) {
            hipMemcpyAsync(&A_ds[i], A_h + i*segmentLen, sizeof(float)*VecSize, hipMemcpyHostToDevice, streams[i]);
            hipMemcpyAsync(&B_ds[i], B_h + i*segmentLen, sizeof(float)*VecSize, hipMemcpyHostToDevice, streams[i]);
            
            // basicSgemmStream(matArow/numStream, matArow/numStream, matArow/numStream, A_ds[Offset], B_ds[Offset], C_ds[Offset], streams[i]);
            basicSgemmStream(matArow,matArow,matArow, A_ds[i], B_d, C_ds[i], streams[i]);
        }
        else {
            hipMemcpyAsync(&A_ds[i], A_h + i*segmentLen, sizeof(float)*(segmentLen + VecSize % numStream), hipMemcpyHostToDevice, streams[i]);
            hipMemcpyAsync(&A_ds[i], B_h + i*segmentLen, sizeof(float)*(segmentLen + VecSize % numStream), hipMemcpyHostToDevice, streams[i]);
            
            // basicSgemmStream(matArow/numStream, matArow/numStream, matArow/numStream, A_ds[Offset], B_ds[Offset], C_ds[Offset], streams[i]);
            basicSgemmStream(matArow,matArow,matArow, A_ds[i], B_d, C_ds[i], streams[i]);
        }
        // else
        // {
        //     hipMemcpyAsync(A_d[i], A_h + i*segmentLen, sizeof(float)*(segmentLen + VecSize % numStream), hipMemcpyHostToDevice, streams[i]);
        //     hipMemcpyAsync(B_d[i], B_h + i*segmentLen, sizeof(float)*(segmentLen + VecSize % numStream), hipMemcpyHostToDevice, streams[i]);
        // }
        // hipMemcpyAsync(&C_h[Offset], &C_d[Offset], sizeof(float)*segmentLen, hipMemcpyDeviceToHost, streams[i]);
        // hipStreamSynchronize(streams[i]);
    }

    for (int i = 0; i < numStream; i++) {
        int Offset = i * segmentLen;
        hipStreamSynchronize(streams[i]);
        // hipMemcpyAsync(&C_h[Offset], &C_d[Offset], sizeof(float)*(segmentLen + VecSize % numStream), hipMemcpyDeviceToHost, streams[i]);
        if (i != numStream-1) {

            hipMemcpyAsync(C_h + i*segmentLen, C_ds[i], sizeof(float)*VecSize, hipMemcpyDeviceToHost, streams[i]);
        } else {
            hipMemcpyAsync(C_h + i*segmentLen, C_ds[i], sizeof(float)*(segmentLen + VecSize % numStream), hipMemcpyDeviceToHost, streams[i]);
        }
    }





    /*************************************************************************/
    
    // hipDeviceSynchronize();
    // stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel using standard sgemm interface ---------------------------
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);


    // basicSgemm(matArow, matBcol, matBrow, A_d, B_d, C_d);
    // for (int i = 0; i < numStream; i++)
    // {
    //     if (i != numStream-1)
    //     {
    //         basicSgemmStream(matArow/numStream, matArow/numStream, matArow/numStream, A_ds[i], B_ds[i], C_ds[i], streams[i]);
    //     }
    //     else
    //     {
    //         basicSgemmStream(matArow/numStream, matArow/numStream + VecSize % numStream,matArow/numStream + VecSize % numStream,A_ds[i], B_ds[i], C_ds[i], streams[i]);
    //     }
    //     hipStreamSynchronize(streams[i]);
    // }










    // cuda_ret = hipDeviceSynchronize();
    // if(cuda_ret != hipSuccess) printf("Unable to launch kernel");
    // stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------
    printf("Copying data from device to host..."); fflush(stdout);
    // startTime(&timer);

    /*************************************************************************/
    //INSERT CODE HERE
    // hipMemcpy(C_h, C_d, sizeof(float) * C_sz, hipMemcpyDeviceToHost);	


    // for (int i = 0; i < numStream; i++)
    // {
    //     if (i != numStream-1)
    //     {
    //         hipMemcpyAsync(C_h , C_ds[i], sizeof(float)*segmentLen, hipMemcpyDeviceToHost, streams[i]);
    //     }
    //     else
    //     {
    //         hipMemcpyAsync(C_h , C_ds[i], sizeof(float)*(segmentLen + VecSize % numStream), hipMemcpyDeviceToHost, streams[i]);
    //     }
    // }









    /*************************************************************************/

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    // printf(C_h, "c_h");fflush(stdout);

    verify(A_h, B_h, C_h, matArow, matAcol, matBcol);


    // Free memory ------------------------------------------------------------
    // free(A_h);
    // free(B_h);
    // free(C_h);

    hipHostFree(A_h);
    hipHostFree(B_h);
    hipHostFree(C_h);
    /*************************************************************************/
    //INSERT CODE HERE
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    for (int i = 0; i < numStream; i++)
    {
        // hipFree(A_ds[i]);
        // hipFree(B_ds[i]);
        // hipFree(C_ds[i]);
        hipStreamDestroy(streams[i]);
    }
    /*************************************************************************/
;
    return 0;
}

